#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>

constexpr auto threads_perblock = 1024;

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        std::cout << hipGetErrorString(err) << "  " << file << "  " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

inline __device__ float fminf(float a, float b)
{
    return a < b ? a : b;
}

inline __device__ float fmaxf(float a, float b)
{
    return a > b ? a : b;
}

inline __device__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

__device__ void hsv2rgb_gpu(int h, int s, int v, int *r, int *g, int *b)
{
    if (h < 0)
        h += (1 - h / 360) * 360;
    if (360 < h)
        h %= 360;
    auto h1 = (h * 4096 + 50) / 120;
    auto s1 = (s * 4096 + 50) / 100;
    auto v1 = (v * 4096 + 50) / 100;
    auto h2 = h1 % 4096;
    auto a1{0}, a2{0};
    if (h2 < 2048)
    {
        a1 = (4096 - (2048 - h2) * s1 / 2048) * v1 / 4096;
        a2 = v1;
    }
    else
    {
        a2 = (4096 - (h2 - 2048) * s1 / 2048) * v1 / 4096;
        a1 = v1;
    }

    auto b1 = clamp((a2 * 255 + 2048) / 4096, 0, 255);
    auto b2 = clamp((a1 * 255 + 2048) / 4096, 0, 255);
    auto b3 = clamp(((4096 - s1) * v1 / 4096 * 255 + 2048) / 4096, 0, 255);

    switch (h1 / 4096)
    {
    case 1:
        *g = b1;
        *b = b2;
        *r = b3;
        break;
    case 2:
        *b = b1;
        *r = b2;
        *g = b3;
        break;
    default:
        *r = b1;
        *g = b2;
        *b = b3;
        break;
    }
}

__device__ int julia_cuda(int x, int y, int view_size)
{
    const auto scale = 1.5f;
    auto jx = scale * static_cast<float>(view_size / 2 - x) / (view_size / 2);
    auto jy = scale * static_cast<float>(view_size / 2 - y) / (view_size / 2);

    thrust::complex<float> c(-0.8f, 0.156f);
    thrust::complex<float> z(jx, jy);

    auto i = 0;
    for (i = 0; i < 360; i++)
    {
        z = z * z + c;
        if (thrust::norm(z) > 1000)
            break;
    }

    return i;
}

__global__ void kernel(unsigned char *ptr, int view_size)
{
    auto r{0}, g{0}, b{0};
    auto x = blockIdx.x;
    auto y = threadIdx.x;
    auto offset = x + y * gridDim.x;
    auto value = julia_cuda(x, y, view_size);

    hsv2rgb_gpu(value, 100, 100, &r, &g, &b);

    if (value >= 0 && value <= 20)
    {
        ptr[offset * 4 + 0] = 255;
        ptr[offset * 4 + 1] = 255;
        ptr[offset * 4 + 2] = 255;
        ptr[offset * 4 + 3] = 255;
    }
    else
    {
        ptr[offset * 4 + 0] = r;
        ptr[offset * 4 + 1] = g;
        ptr[offset * 4 + 2] = b;
        ptr[offset * 4 + 3] = 255;
    }
}

extern "C" unsigned char *JuliaGPU(std::size_t size, int view_size)
{
    unsigned char *ptr_gpu;

    HANDLE_ERROR(hipMalloc((void **)&ptr_gpu, size));

    auto *ptr = new unsigned char[size];
    HANDLE_ERROR(hipMemcpy(ptr_gpu, ptr, size, hipMemcpyHostToDevice));

    auto blocks_per_grid = ((view_size * view_size) + threads_perblock - 1) / threads_perblock;
    std::cout << "CUDA kernel [" << blocks_per_grid << "] blocks [" << threads_perblock << "] threads" << std::endl;
    kernel<<<blocks_per_grid, threads_perblock>>>(ptr_gpu, view_size);

    HANDLE_ERROR(hipMemcpy(ptr, ptr_gpu, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(ptr_gpu));
    return ptr;
}
